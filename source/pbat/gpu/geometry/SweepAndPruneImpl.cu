#include "hip/hip_runtime.h"
#include "SweepAndPruneImpl.cuh"

#include <cuda/atomic>
#include <cuda/std/cmath>
#include <exception>
#include <string>
#include <thrust/async/copy.h>
#include <thrust/async/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <tuple>

namespace pbat {
namespace gpu {
namespace geometry {

SweepAndPruneImpl::SweepAndPruneImpl(std::size_t nPrimitives, std::size_t nOverlaps)
    : binds(nPrimitives),
      sinds(
          {thrust::device_vector<GpuIndex>(nPrimitives),
           thrust::device_vector<GpuIndex>(nPrimitives),
           thrust::device_vector<GpuIndex>(nPrimitives),
           thrust::device_vector<GpuIndex>(nPrimitives)}),
      b({thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives)}),
      e({thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives)}),
      mu(3, 0.f),
      sigma(3, 0.f),
      no(1),
      o(nOverlaps)
{
}

struct FComputeAabb
{
    __device__ void operator()(int s)
    {
        for (auto d = 0; d < 3; ++d)
        {
            b[d][s] = x[d][inds[0][s]];
            e[d][s] = x[d][inds[0][s]];
            for (auto m = 1; m < nSimplexVertices; ++m)
            {
                b[d][s] = cuda::std::fminf(b[d][s], x[d][inds[m][s]]);
                e[d][s] = cuda::std::fmaxf(b[d][s], x[d][inds[m][s]]);
            }
            b[d][s] -= r;
            e[d][s] += r;
        }
    }

    std::array<GpuScalar const*, 3> x;
    std::array<GpuIndex*, 4> inds;
    int nSimplexVertices;
    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    GpuScalar r = 0.;
};

struct FComputeMean
{
    __device__ void operator()(int s)
    {
        cuda::atomic_ref<GpuScalar, cuda::thread_scope_device> amu[3] = {mu[0], mu[1], mu[2]};
        for (auto d = 0; d < 3; ++d)
        {
            amu[d] += (b[d][s] + e[d][s]) / (2.f * static_cast<GpuScalar>(nBoxes));
        }
    }

    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    GpuScalar* mu;
    GpuIndex nBoxes;
};

struct FComputeVariance
{
    __device__ void operator()(int s)
    {
        cuda::atomic_ref<GpuScalar, cuda::thread_scope_device> asigma[3] = {
            sigma[0],
            sigma[1],
            sigma[2]};
        for (auto d = 0; d < 3; ++d)
        {
            GpuScalar const cd = (b[d][s] + e[d][s]) / 2.f;
            GpuScalar const dx = cd - mu[d];
            asigma[d] += dx * dx / static_cast<GpuScalar>(nBoxes);
        }
    }

    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    GpuScalar* mu;
    GpuScalar* sigma;
    GpuIndex nBoxes;
};

struct FSweep
{
    /**
     * @brief If (si,sj) are from the same simplex set, or if (si,sj) share a common vertex, they
     * should not be considered for overlap testing.
     * @param sinds Simplex vertex indices in both sets
     * @param nSimplices Number of simplices in each simplex set
     * @param si Index of first simplex in pair to test
     * @param sj Index of second simplex in pair to test
     * @return
     */
    __device__ bool AreSimplicesOverlapCandidates(GpuIndex si, GpuIndex sj) const
    {
        if ((binds[si] < nSimplices[0]) == (binds[sj] < nSimplices[0]))
            return false;
        for (auto i = 0; i < sinds.size(); ++i)
            for (auto j = 0; j < sinds.size(); ++j)
                if (sinds[i][si] == sinds[j][sj])
                    return false;
        return true;
    }

    __device__ bool AreSimplexCandidatesOverlapping(GpuIndex si, GpuIndex sj) const
    {
        return (e[axis[0]][si] >= b[axis[0]][sj] or b[axis[0]][si] <= e[axis[0]][sj]) and
               (e[axis[1]][si] >= b[axis[1]][sj] or b[axis[1]][si] <= e[axis[1]][sj]);
    }

    __device__ void operator()(GpuIndex si)
    {
        cuda::atomic_ref<GpuIndex, cuda::thread_scope_device> ano{*no};
        bool const bSwap = binds[si] >= nSimplices[0];
        for (auto sj = si + 1; (sj < nBoxes) and (e[saxis][si] >= b[saxis][sj]); ++sj)
        {
            if (not AreSimplicesOverlapCandidates(si, sj))
                continue;
            if (not AreSimplexCandidatesOverlapping(si, sj))
                continue;
            GpuIndex k = ano++;
            if (k >= nOverlapCapacity)
                break;

            if (not bSwap)
                o[k] = {binds[si], binds[sj] - nSimplices[0]};
            else
                o[k] = {binds[sj], binds[si] - nSimplices[0]};
        }
    }

    GpuIndex* binds;
    std::array<GpuIndex*, 4> sinds;
    std::array<GpuIndex, 2> nSimplices;
    std::array<GpuScalar*, 3> b, e;
    GpuIndex saxis;
    std::array<GpuIndex, 2> axis;
    GpuIndex* no;
    SweepAndPruneImpl::OverlapType* o;
    GpuIndex nBoxes;
    GpuIndex nOverlapCapacity;
};

void SweepAndPruneImpl::SortAndSweep(
    Points const& P,
    Simplices const& S1,
    Simplices const& S2,
    GpuScalar expansion)
{
    auto const nBoxes = S1.NumberOfSimplices() + S2.NumberOfSimplices();
    if (NumberOfAllocatedBoxes() < nBoxes)
    {
        std::string const what = "Allocated memory for " +
                                 std::to_string(NumberOfAllocatedBoxes()) +
                                 " boxes, but received " + std::to_string(nBoxes) + " simplices.";
        throw std::invalid_argument(what);
    }

    // 0. Preprocess internal data
    mu[0] = mu[1] = mu[2] = 0.f;
    sigma[0] = sigma[1] = sigma[2] = 0.f;
    no[0]                          = 0;
    thrust::sequence(thrust::device, binds.begin(), binds.end());
    auto const boxesBegin = thrust::make_counting_iterator(0);
    auto const boxesEnd   = thrust::make_counting_iterator(nBoxes);

    // Convert thrust pointers to raw device pointers, since we need to store them in our functors,
    // and can't store/access host memory there.
    std::array<GpuScalar*, 3> bRaw{
        thrust::raw_pointer_cast(b[0].data()),
        thrust::raw_pointer_cast(b[1].data()),
        thrust::raw_pointer_cast(b[2].data())};
    std::array<GpuScalar*, 3> eRaw{
        thrust::raw_pointer_cast(e[0].data()),
        thrust::raw_pointer_cast(e[1].data()),
        thrust::raw_pointer_cast(e[2].data())};
    std::array<GpuIndex*, 4> sindsRaw{
        thrust::raw_pointer_cast(sinds[0].data()),
        thrust::raw_pointer_cast(sinds[1].data()),
        thrust::raw_pointer_cast(sinds[2].data()),
        thrust::raw_pointer_cast(sinds[3].data())};

    // 1. Compute bounding boxes of S1 and S2
    std::array<thrust::device_event, 8> sindsCopyEvents{};
    for (auto m = 0; m < 4; ++m)
    {
        sindsCopyEvents[m * 2ULL] = thrust::async::copy(
            thrust::device,
            S1.inds[m].begin(),
            S1.inds[m].end(),
            sinds[m].begin());
        sindsCopyEvents[m * 2ULL + 1ULL] = thrust::async::copy(
            thrust::device,
            S2.inds[m].begin(),
            S2.inds[m].end(),
            sinds[m].begin() + S1.NumberOfSimplices());
    }
    auto computeAabbExecutionPolicy = thrust::device.after(
        sindsCopyEvents[0],
        sindsCopyEvents[1],
        sindsCopyEvents[2],
        sindsCopyEvents[3],
        sindsCopyEvents[4],
        sindsCopyEvents[5],
        sindsCopyEvents[6],
        sindsCopyEvents[7]);
    thrust::device_event computeAabbEvent;
    computeAabbEvent = thrust::async::for_each(
        computeAabbExecutionPolicy,
        boxesBegin,
        thrust::make_counting_iterator(S1.NumberOfSimplices()),
        FComputeAabb{P.Raw(), sindsRaw, static_cast<int>(S1.eSimplexType), bRaw, eRaw, expansion});
    computeAabbEvent = thrust::async::for_each(
        thrust::device.after(computeAabbEvent),
        thrust::make_counting_iterator(S1.NumberOfSimplices()),
        boxesEnd,
        FComputeAabb{P.Raw(), sindsRaw, static_cast<int>(S2.eSimplexType), bRaw, eRaw, expansion});

    // 2. Compute mean and variance of bounding box centers
    auto muRaw = thrust::raw_pointer_cast(mu.data());
    FComputeMean fComputeMean{bRaw, eRaw, muRaw, nBoxes};
    thrust::device_event computeMeanEvent = thrust::async::for_each(
        thrust::device.after(computeAabbEvent),
        boxesBegin,
        boxesEnd,
        fComputeMean);
    auto sigmaRaw = thrust::raw_pointer_cast(sigma.data());
    FComputeVariance fComputeVariance{bRaw, eRaw, muRaw, sigmaRaw, nBoxes};
    thrust::device_event computeVarianceEvent = thrust::async::for_each(
        thrust::device.after(computeMeanEvent),
        boxesBegin,
        boxesEnd,
        fComputeVariance);
    computeVarianceEvent.wait();

    // 3. Sort bounding boxes along largest variance axis
    GpuIndex const saxis =
        (sigma[0] > sigma[1]) ? (sigma[0] > sigma[2] ? 0 : 2) : (sigma[1] > sigma[2] ? 1 : 2);
    std::array<GpuIndex, 2> const axis = {(saxis + 1) % 3, (saxis + 2) % 3};
    thrust::sort_by_key(
        thrust::device,
        b[saxis].begin(),
        b[saxis].end(),
        thrust::make_zip_iterator(
            binds.begin(),
            sinds[0].begin(),
            sinds[1].begin(),
            sinds[2].begin(),
            sinds[3].begin(),
            b[axis[0]].begin(),
            b[axis[1]].begin(),
            e[axis[0]].begin(),
            e[axis[1]].begin()));

    // 4. Sweep to find overlaps
    FSweep fSweep{
        thrust::raw_pointer_cast(binds.data()),
        sindsRaw,
        {S1.NumberOfSimplices(), S2.NumberOfSimplices()},
        bRaw,
        eRaw,
        saxis,
        axis,
        thrust::raw_pointer_cast(no.data()),
        thrust::raw_pointer_cast(o.data()),
        nBoxes,
        static_cast<GpuIndex>(o.size())};
    thrust::for_each(thrust::device, boxesBegin, boxesEnd, fSweep);
}

std::size_t SweepAndPruneImpl::NumberOfAllocatedBoxes() const
{
    return binds.size();
}

std::size_t SweepAndPruneImpl::NumberOfAllocatedOverlaps() const
{
    return o.size();
}

} // namespace geometry
} // namespace gpu
} // namespace pbat

#include <doctest/doctest.h>

TEST_CASE("[gpu][geometry] Sweep and prune")
{
    using namespace pbat;
    MatrixX V(3, 7);
    IndexMatrixX E1(2, 3);
    IndexMatrixX F2(3, 1);
    // clang-format off
    V << 0.,  1. ,  2. ,  3. , 0.,  2. ,  0.,
         0.,  0.1,  0.2,  0.3, 0.,  0.1,  0.,
         0., 10. , 20. , 30. , 0., 10. ,  0.;
    E1 << 1, 0, 2,
          2, 1, 3;
    F2 << 4,
          5,
          6;
    // clang-format on
    gpu::geometry::Points P(V);
    gpu::geometry::Simplices S1(E1);
    gpu::geometry::Simplices S2(F2);

    gpu::geometry::SweepAndPruneImpl stq(4, 2);
    stq.SortAndSweep(P, S1, S2);
}
