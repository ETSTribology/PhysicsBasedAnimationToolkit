#include "hip/hip_runtime.h"
#include "SweepAndPruneImpl.cuh"

#include <cuda/atomic>
#include <cuda/std/cmath>
#include <exception>
#include <string>
#include <thrust/async/copy.h>
#include <thrust/async/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

namespace pbat {
namespace gpu {
namespace geometry {

SweepAndPruneImpl::SweepAndPruneImpl(std::size_t nPrimitives, std::size_t nOverlaps)
    : binds(nPrimitives),
      b({thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives)}),
      e({thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives),
         thrust::device_vector<GpuScalar>(nPrimitives)}),
      mu(3, 0.f),
      sigma(3, 0.f),
      no(1),
      o(nOverlaps)
{
}

struct FComputeAabb
{
    FComputeAabb(
        std::array<thrust::device_ptr<GpuScalar const>, 3> xIn,
        Simplices const& SIn,
        std::array<thrust::device_ptr<GpuScalar>, 3> bIn,
        std::array<thrust::device_ptr<GpuScalar>, 3> eIn)
        : x({thrust::raw_pointer_cast(xIn[0]),
             thrust::raw_pointer_cast(xIn[1]),
             thrust::raw_pointer_cast(xIn[2])}),
          nSimplexVertices(static_cast<int>(SIn.eSimplexType)),
          inds(thrust::raw_pointer_cast(SIn.inds.data())),
          b({thrust::raw_pointer_cast(bIn[0]),
             thrust::raw_pointer_cast(bIn[1]),
             thrust::raw_pointer_cast(bIn[2])}),
          e({thrust::raw_pointer_cast(eIn[0]),
             thrust::raw_pointer_cast(eIn[1]),
             thrust::raw_pointer_cast(eIn[2])})
    {
    }

    __device__ void operator()(int s)
    {
        auto const begin = s * nSimplexVertices;
        auto const end   = begin + nSimplexVertices;
        for (auto d = 0; d < 3; ++d)
        {
            b[d][s] = x[d][inds[begin]];
            e[d][s] = x[d][inds[begin]];
            for (auto i = begin + 1; i < end; ++i)
            {
                b[d][s] = cuda::std::fminf(b[d][s], x[d][inds[i]]);
                e[d][s] = cuda::std::fmaxf(b[d][s], x[d][inds[i]]);
            }
            // TODO: Add some inflation to bounding box endpoints to support activation distance
        }
    }

    std::array<GpuScalar const*, 3> x;
    int nSimplexVertices;
    GpuIndex const* inds;
    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
};

struct FComputeMean
{
    FComputeMean(
        std::array<thrust::device_ptr<GpuScalar const>, 3> bIn,
        std::array<thrust::device_ptr<GpuScalar const>, 3> eIn,
        thrust::device_ptr<GpuScalar> muIn,
        GpuIndex nBoxesIn)
        : b({thrust::raw_pointer_cast(bIn[0]),
             thrust::raw_pointer_cast(bIn[1]),
             thrust::raw_pointer_cast(bIn[2])}),
          e({thrust::raw_pointer_cast(eIn[0]),
             thrust::raw_pointer_cast(eIn[1]),
             thrust::raw_pointer_cast(eIn[2])}),
          mu(thrust::raw_pointer_cast(muIn)),
          nBoxes(nBoxesIn)
    {
    }

    __device__ void operator()(int s)
    {
        cuda::atomic_ref<GpuScalar, cuda::thread_scope_device> amu[3] = {mu[0], mu[1], mu[2]};
        for (auto d = 0; d < 3; ++d)
        {
            amu[d] += (b[d][s] + e[d][s]) / (2.f * static_cast<GpuScalar>(nBoxes));
        }
    }

    std::array<GpuScalar const*, 3> b;
    std::array<GpuScalar const*, 3> e;
    GpuScalar* mu;
    GpuIndex nBoxes;
};

struct FComputeVariance
{
    FComputeVariance(
        std::array<thrust::device_ptr<GpuScalar const>, 3> bIn,
        std::array<thrust::device_ptr<GpuScalar const>, 3> eIn,
        thrust::device_ptr<GpuScalar> muIn,
        thrust::device_ptr<GpuScalar> sigmaIn,
        GpuIndex nBoxesIn)
        : b({thrust::raw_pointer_cast(bIn[0]),
             thrust::raw_pointer_cast(bIn[1]),
             thrust::raw_pointer_cast(bIn[2])}),
          e({thrust::raw_pointer_cast(eIn[0]),
             thrust::raw_pointer_cast(eIn[1]),
             thrust::raw_pointer_cast(eIn[2])}),
          mu(muIn),
          sigma(thrust::raw_pointer_cast(sigmaIn)),
          nBoxes(nBoxesIn)
    {
    }

    __device__ void operator()(int s)
    {
        cuda::atomic_ref<GpuScalar, cuda::thread_scope_device> asigma[3] = {
            sigma[0],
            sigma[1],
            sigma[2]};
        for (auto d = 0; d < 3; ++d)
        {
            GpuScalar const cd = (b[d][s] + e[d][s]) / 2.f;
            GpuScalar const dx = cd - mu[d];
            asigma[d] += dx * dx / static_cast<GpuScalar>(nBoxes);
        }
    }

    std::array<GpuScalar const*, 3> b;
    std::array<GpuScalar const*, 3> e;
    thrust::device_ptr<GpuScalar const> mu;
    GpuScalar* sigma;
    GpuIndex nBoxes;
};

__device__ bool AreSimplicesAdjacent(
    GpuIndex* sinds1,
    int nSimplexVertices1,
    GpuIndex s1,
    GpuIndex* sinds2,
    int nSimplexVertices2,
    GpuIndex s2)
{
    auto const begin1 = s1 * nSimplexVertices1;
    auto const end1   = begin1 + nSimplexVertices1;
    auto const begin2 = s2 * nSimplexVertices2;
    auto const end2   = begin2 + nSimplexVertices2;
    bool bAreAdjacent{false};
    for (auto i = begin1; (i < end1) and not bAreAdjacent; ++i)
    {
        for (auto j = begin2; j < end2; ++j)
        {
            if (sinds1[i] == sinds2[j])
            {
                bAreAdjacent = true;
                break;
            }
        }
    }
    return bAreAdjacent;
}

//__global__ void SweepImpl(
//    GpuIndex* binds,
//    GpuScalar* b[3],
//    GpuScalar* e[3],
//    GpuIndex nSimplices1,
//    GpuIndex* sinds1,
//    int nSimplexVertices1,
//    GpuIndex* sinds2,
//    int nSimplexVertices2,
//    GpuIndex saxis,
//    GpuIndex axis[2],
//    GpuIndex nBoxes,
//    GpuIndex* no,
//    cuda::std::pair<GpuIndex, GpuIndex>* o)
//{
//    cuda::atomic_ref<GpuIndex, cuda::thread_scope_device> ano{*no};
//    GpuIndex const t = threadIdx.x + blockIdx.x * blockDim.x;
//    if (t >= nBoxes)
//        return;
//
//    GpuIndex tp = t + 1;
//    while (tp >= nBoxes and e[saxis][t] > b[saxis][tp])
//    {
//        GpuIndex const s1                         = binds[t];
//        bool const bAreSimplicesFromDifferentSets = binds[tp] > nSimplices1;
//        if (bAreSimplicesFromDifferentSets)
//        {
//            GpuIndex const s2 =
//                bAreSimplicesFromDifferentSets ? binds[tp] - nSimplices1 : binds[tp];
//            bool const bAreAdjacent =
//                AreSimplicesAdjacent(sinds1, nSimplexVertices1, s1, sinds2, nSimplexVertices2,
//                s2);
//            bool const bAreOverlapping =
//                e[axis[0]][t] > b[axis[0]][tp] and e[axis[1]][t] > b[axis[1]][tp];
//            if (bAreOverlapping and not bAreAdjacent)
//            {
//                GpuIndex const oid = ano++;
//                o[oid]             = {s1, s2};
//            }
//        }
//        ++tp;
//    }
//}

void SweepAndPruneImpl::SortAndSweep(Points const& P, Simplices const& S1, Simplices const& S2)
{
    auto const nSimplices1 = S1.NumberOfSimplices();
    auto const nSimplices2 = S2.NumberOfSimplices();
    auto const nBoxes      = nSimplices1 + nSimplices2;
    if (NumberOfAllocatedBoxes() < nBoxes)
    {
        std::string const what = "Allocated memory for " +
                                 std::to_string(NumberOfAllocatedBoxes()) +
                                 " boxes, but received " + std::to_string(nBoxes) + " simplices.";
        throw std::invalid_argument(what);
    }

    // 1. Compute bounding boxes of S1 and S2
    thrust::device_event computeAabbEvent1 = thrust::async::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(nSimplices1),
        FComputeAabb(
            {P.x.data(), P.y.data(), P.z.data()},
            S1,
            {b[0].data(), b[1].data(), b[2].data()},
            {e[0].data(), e[1].data(), e[2].data()}));
    thrust::device_event computeAabbEvent2 = thrust::async::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(nSimplices2),
        FComputeAabb(
            {P.x.data(), P.y.data(), P.z.data()},
            S2,
            {b[0].data() + nSimplices1, b[1].data() + nSimplices1, b[2].data() + nSimplices1},
            {e[0].data() + nSimplices1, e[1].data() + nSimplices1, e[2].data() + nSimplices1}));

    // 2. Compute mean and variance of bounding box centers
    auto const boxesBegin = thrust::make_counting_iterator(0);
    auto const boxesEnd   = thrust::make_counting_iterator(nBoxes);
    thrust::fill(mu.begin(), mu.end(), 0.f);
    thrust::fill(sigma.begin(), sigma.end(), 0.f);
    FComputeMean fComputeMean(
        {b[0].data(), b[1].data(), b[2].data()},
        {e[0].data(), e[1].data(), e[2].data()},
        mu.data(),
        nBoxes);
    thrust::device_event computeMeanEvent = thrust::async::for_each(
        thrust::device.after(computeAabbEvent1, computeAabbEvent2),
        boxesBegin,
        boxesEnd,
        fComputeMean);
    FComputeVariance fComputeVariance(
        {b[0].data(), b[1].data(), b[2].data()},
        {e[0].data(), e[1].data(), e[2].data()},
        mu.data(),
        sigma.data(),
        nBoxes);
    thrust::device_event computeVarianceEvent = thrust::async::for_each(
        thrust::device.after(computeMeanEvent),
        boxesBegin,
        boxesEnd,
        fComputeVariance);
    computeVarianceEvent.wait();

    // 3. Sort bounding boxes along largest variance axis
    GpuIndex const saxis =
        (sigma[0] > sigma[1]) ? (sigma[0] > sigma[2] ? 0 : 2) : (sigma[1] > sigma[2] ? 1 : 2);
    thrust::sequence(thrust::device, binds.begin(), binds.end());
    GpuIndex const axis[2] = {(saxis + 1) % 3, (saxis + 2) % 3};
    thrust::sort_by_key(
        thrust::device,
        b[saxis].begin(),
        b[saxis].end(),
        thrust::make_zip_iterator(
            binds.begin(),
            b[axis[0]].begin(),
            b[axis[1]].begin(),
            e[axis[0]].begin(),
            e[axis[1]].begin()));

    // 4. Sweep to find overlaps
    thrust::fill(no.begin(), no.end(), 0);
    auto const nThreadsPerBlock = 32;
    auto const nBlocks          = (nBoxes - 1) / nThreadsPerBlock + 1;
    // SweepImpl<<<nBlocks, nThreadsPerBlock>>>();
    //  TODO: ...
}

std::size_t SweepAndPruneImpl::NumberOfAllocatedBoxes() const
{
    return binds.size();
}

std::size_t SweepAndPruneImpl::NumberOfAllocatedOverlaps() const
{
    return o.size();
}

} // namespace geometry
} // namespace gpu
} // namespace pbat

#include <doctest/doctest.h>

TEST_CASE("[gpu][geometry] Sweep and tiniest queue")
{
    using namespace pbat;
    MatrixX V(3, 4);
    // clang-format off
    V << 0., 1., 2., 3.,
         0., 0., 0., 0.,
         0., 10., 20., 30.;
    // clang-format on
    IndexMatrixX E(2, 3);
    // clang-format off
    E << 1, 0, 2,
         2, 1, 3;
    // clang-format on
    gpu::geometry::Points P(V);
    gpu::geometry::Simplices S(E);

    gpu::geometry::SweepAndPruneImpl stq(3, 2);
    // stq.SortAndSweep(P, S);
}
