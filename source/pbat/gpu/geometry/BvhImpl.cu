#include "hip/hip_runtime.h"
// clang-format off
#include "pbat/gpu/DisableWarnings.h"
// clang-format on

#include "BvhImpl.cuh"

#include <cuda/atomic>
#include <exception>
#include <string>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <type_traits>

namespace pbat {
namespace gpu {
namespace geometry {

struct FLeafBoundingBoxes
{
    __device__ void operator()(int s)
    {
        using namespace cuda::std;
        for (auto d = 0; d < 3; ++d)
        {
            auto bs  = leafBegin + s;
            b[d][bs] = x[d][inds[0][s]];
            e[d][bs] = x[d][inds[0][s]];
            for (auto m = 1; m < nSimplexVertices; ++m)
            {
                b[d][bs] = fminf(b[d][bs], x[d][inds[m][s]]);
                e[d][bs] = fmaxf(e[d][bs], x[d][inds[m][s]]);
            }
            b[d][bs] -= r;
            e[d][bs] += r;
        }
    }

    std::array<GpuScalar const*, 3> x;
    std::array<GpuIndex const*, 4> inds;
    int nSimplexVertices;
    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    GpuIndex leafBegin;
    GpuScalar r;
};

struct FComputeMortonCode
{
    using MortonCodeType = typename BvhImpl::MortonCodeType;

    // Expands a 10-bit integer into 30 bits
    // by inserting 2 zeros after each bit.
    __device__ MortonCodeType ExpandBits(MortonCodeType v)
    {
        v = (v * 0x00010001u) & 0xFF0000FFu;
        v = (v * 0x00000101u) & 0x0F00F00Fu;
        v = (v * 0x00000011u) & 0xC30C30C3u;
        v = (v * 0x00000005u) & 0x49249249u;
        return v;
    }

    // Calculates a 30-bit Morton code for the
    // given 3D point located within the unit cube [0,1].
    __device__ MortonCodeType Morton3D(std::array<GpuScalar, 3> x)
    {
        using namespace cuda::std;
        x[0]              = fminf(fmaxf(x[0] * 1024.0f, 0.0f), 1023.0f);
        x[1]              = fminf(fmaxf(x[1] * 1024.0f, 0.0f), 1023.0f);
        x[2]              = fminf(fmaxf(x[2] * 1024.0f, 0.0f), 1023.0f);
        MortonCodeType xx = ExpandBits(static_cast<MortonCodeType>(x[0]));
        MortonCodeType yy = ExpandBits(static_cast<MortonCodeType>(x[1]));
        MortonCodeType zz = ExpandBits(static_cast<MortonCodeType>(x[2]));
        return xx * 4 + yy * 2 + zz;
    }

    __device__ void operator()(int s)
    {
        auto const bs = leafBegin + s;
        // Compute Morton code of the centroid of the bounding box of simplex s
        std::array<GpuScalar, 3> c{0.f, 0.f, 0.f};
        for (auto d = 0; d < 3; ++d)
            c[d] += GpuScalar{0.5} * (b[d][bs] + e[d][bs]);
        morton[s] = Morton3D(c);
    }

    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    MortonCodeType* morton;
    GpuIndex leafBegin;
};

struct FGenerateHierarchy
{
    using MortonCodeType = typename BvhImpl::MortonCodeType;

    struct Range
    {
        GpuIndex i, j, l;
        int d;
    };

    __device__ int Delta(GpuIndex i, GpuIndex j) const
    {
        if (j < 0 or j >= n)
            return -1;
        if (i == j)
            return __clz(i ^ j);
        return __clz(morton[i] ^ morton[j]);
    }

    __device__ Range DetermineRange(GpuIndex i) const
    {
        // Compute range direction
        int const d = (Delta(i, i + 1) - Delta(i, i - 1)) >= 0;
        // Lower bound on length of internal node i's common prefix
        int const dmin = Delta(i, i - d);
        // Compute conservative upper bound on the range's size
        GpuIndex lmax{2};
        while (Delta(i, i + lmax * d) > dmin)
            lmax <<= 1;
        // Binary search in the "inflated" range for the actual end (or start) of internal node i's
        // range, considering that i is its start (or end).
        GpuIndex l{0};
        do
        {
            lmax >>= 1;
            if (Delta(i, i + (l + lmax) * d) > dmin)
                l += lmax;
        } while (lmax > 1);
        GpuIndex j = i + l * d;
        return Range{i, j, l, d};
    }

    __device__ GpuIndex FindSplit(Range R) const
    {
        // Identical Morton codes => split the range in the middle.
        if (morton[R.i] == morton[R.j])
            return (R.i + R.j) >> 1;

        // Calculate the number of highest bits that are the same
        // for all objects.
        int const dnode = Delta(R.i, R.j);

        // Use binary search to find where the next bit differs.
        // Specifically, we are looking for the highest object that
        // shares more than dnode bits with the first one.
        GpuIndex s{0};
        do
        {
            R.l = (R.l + 1) >> 1;
            if (Delta(R.i, R.i + (s + R.l) * R.d) > dnode)
                s += R.l;
        } while (R.l > 1);
        GpuIndex const gamma = R.i + s * R.d + min(R.d, 0);
        return gamma;
    }

    __device__ void operator()(auto i)
    {
        // Find out which range of objects the node corresponds to.
        Range R = DetermineRange(i);
        // Determine where to split the range.
        GpuIndex gamma = FindSplit(R);
        // Select left+right child
        GpuIndex lc = (min(R.i, R.j) == gamma) ? leafBegin + gamma : gamma;
        GpuIndex rc = (max(R.i, R.j) == gamma + 1) ? leafBegin + gamma + 1 : gamma + 1;
        // Record parent-child relationships
        child[0][i] = lc;
        child[1][i] = rc;
        parent[lc]  = i;
        parent[rc]  = i;
    }

    MortonCodeType const* morton;
    std::array<GpuIndex*, 2> child;
    GpuIndex* parent;
    GpuIndex leafBegin;
    GpuIndex n;
};

struct FInternalNodeBoundingBoxes
{
    __device__ void operator()(auto leaf)
    {
        using namespace cuda::std;
        auto p = parent[leaf];
        while (p >= 0)
        {
            cuda::atomic_ref<GpuIndex, cuda::thread_scope_device> ap{visits[p]};
            // The first thread that gets access to the internal node p will terminate,
            // while the second thread visiting p will be allowed to continue execution.
            // This ensures that there is no race condition where a thread can access an
            // internal node too early, i.e. before both children of the internal node
            // have finished computing their bounding boxes.
            if (ap++ == 0)
                break;

            GpuIndex lc = child[0][p];
            GpuIndex rc = child[1][p];
            for (auto d = 0; d < 3; ++d)
            {
                b[d][p] = fminf(b[d][lc], b[d][rc]);
                e[d][p] = fmaxf(e[d][lc], e[d][rc]);
            }
            // Move up the binary tree
            p = parent[p];
        }
    }

    GpuIndex const* parent;
    std::array<GpuIndex*, 2> child;
    std::array<GpuScalar*, 3> b;
    std::array<GpuScalar*, 3> e;
    GpuIndex* visits;
};

BvhImpl::BvhImpl(std::size_t nPrimitives, std::size_t nOverlaps)
    : simplex(nPrimitives),
      morton(nPrimitives),
      child(nPrimitives - 1),
      parent(2 * nPrimitives - 1),
      b(2 * nPrimitives - 1),
      e(2 * nPrimitives - 1),
      visits(nPrimitives - 1),
      no(0),
      o(nOverlaps)
{
    thrust::fill(thrust::device, parent.Data(), parent.Data() + parent.Size(), GpuIndex{-1});
}

void BvhImpl::Build(PointsImpl const& P, SimplicesImpl const& S, GpuScalar expansion)
{
    auto const n = S.NumberOfSimplices();
    if (NumberOfAllocatedBoxes() < n)
    {
        std::string const what = "Allocated memory for " +
                                 std::to_string(NumberOfAllocatedBoxes()) +
                                 " boxes, but received " + std::to_string(n) + " simplices.";
        throw std::invalid_argument(what);
    }

    // 0. Reset intermediate data
    thrust::fill(thrust::device, visits.Raw(), visits.Raw() + visits.Size(), GpuIndex{0});

    // 1. Construct leaf node (i.e. simplex) bounding boxes
    auto const leafBegin = n - 1;
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(n),
        FLeafBoundingBoxes{
            P.x.Raw(),
            S.inds.Raw(),
            static_cast<int>(S.eSimplexType),
            b.Raw(),
            e.Raw(),
            leafBegin,
            expansion});

    // 2. Compute Morton codes for each leaf node (i.e. simplex)
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(n),
        FComputeMortonCode{b.Raw(), e.Raw(), morton.Raw(), leafBegin});

    // 3. Sort simplices based on Morton codes
    thrust::sequence(thrust::device, simplex.Data(), simplex.Data());
    auto zip = thrust::make_zip_iterator(
        b[0].begin() + leafBegin,
        b[1].begin() + leafBegin,
        b[2].begin() + leafBegin,
        e[0].begin() + leafBegin,
        e[1].begin() + leafBegin,
        e[2].begin() + leafBegin,
        simplex.Data());
    // Using a stable sort preserves the initial ordering of simplex indices 0...n-1, resulting in
    // simplices sorted by Morton codes first, and then by simplex index.
    thrust::stable_sort_by_key(thrust::device, morton.Data(), morton.Data() + n, zip);

    // 4. Construct hierarchy
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(0),
        thrust::make_counting_iterator(n - 1),
        FGenerateHierarchy{morton.Raw(), child.Raw(), parent.Raw(), leafBegin, n});

    // 5. Construct internal node bounding boxes
    thrust::for_each(
        thrust::device,
        thrust::make_counting_iterator(n - 1),
        thrust::make_counting_iterator(2 * n - 1),
        FInternalNodeBoundingBoxes{parent.Raw(), child.Raw(), b.Raw(), e.Raw(), visits.Raw()});
}

std::size_t BvhImpl::NumberOfAllocatedBoxes() const
{
    return simplex.Size();
}

} // namespace geometry
} // namespace gpu
} // namespace pbat